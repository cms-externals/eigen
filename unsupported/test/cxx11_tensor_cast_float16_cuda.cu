// This file is part of Eigen, a lightweight C++ template library
// for linear algebra.
//
// Copyright (C) 2016 Benoit Steiner <benoit.steiner.goog@gmail.com>
//
// This Source Code Form is subject to the terms of the Mozilla
// Public License v. 2.0. If a copy of the MPL was not distributed
// with this file, You can obtain one at http://mozilla.org/MPL/2.0/.

#define EIGEN_TEST_NO_LONGDOUBLE
#define EIGEN_TEST_NO_COMPLEX
#define EIGEN_TEST_FUNC cxx11_tensor_cast_float16_cuda
#define EIGEN_DEFAULT_DENSE_INDEX_TYPE int
#define EIGEN_USE_GPU

#if defined __CUDACC_VER__ && __CUDACC_VER__ >= 70500
#include <hip/hip_fp16.h>
#endif
#include "main.h"
#include <unsupported/Eigen/CXX11/Tensor>

using Eigen_tf::Tensor;

void test_cuda_conversion() {
  Eigen_tf::CudaStreamDevice stream;
  Eigen_tf::GpuDevice gpu_device(&stream);
  int num_elem = 101;

  Tensor<float, 1> floats(num_elem);
  floats.setRandom();

  float* d_float = (float*)gpu_device.allocate(num_elem * sizeof(float));
  Eigen_tf::half* d_half = (Eigen_tf::half*)gpu_device.allocate(num_elem * sizeof(Eigen_tf::half));
  float* d_conv = (float*)gpu_device.allocate(num_elem * sizeof(float));

  Eigen_tf::TensorMap<Eigen_tf::Tensor<float, 1>, Eigen_tf::Aligned> gpu_float(
      d_float, num_elem);
  Eigen_tf::TensorMap<Eigen_tf::Tensor<Eigen_tf::half, 1>, Eigen_tf::Aligned> gpu_half(
      d_half, num_elem);
  Eigen_tf::TensorMap<Eigen_tf::Tensor<float, 1>, Eigen_tf::Aligned> gpu_conv(
      d_conv, num_elem);

  gpu_device.memcpyHostToDevice(d_float, floats.data(), num_elem*sizeof(float));

  gpu_half.device(gpu_device) = gpu_float.cast<Eigen_tf::half>();
  gpu_conv.device(gpu_device) = gpu_half.cast<float>();

  Tensor<float, 1> initial(num_elem);
  Tensor<float, 1> final(num_elem);
  gpu_device.memcpyDeviceToHost(initial.data(), d_float, num_elem*sizeof(float));
  gpu_device.memcpyDeviceToHost(final.data(), d_conv, num_elem*sizeof(float));
  gpu_device.synchronize();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(initial(i), final(i));
  }

  gpu_device.deallocate(d_float);
  gpu_device.deallocate(d_half);
  gpu_device.deallocate(d_conv);
}


void test_fallback_conversion() {
  int num_elem = 101;
  Tensor<float, 1> floats(num_elem);
  floats.setRandom();

  Eigen_tf::Tensor<Eigen_tf::half, 1> halfs = floats.cast<Eigen_tf::half>();
  Eigen_tf::Tensor<float, 1> conv = halfs.cast<float>();

  for (int i = 0; i < num_elem; ++i) {
    VERIFY_IS_APPROX(floats(i), conv(i));
  }
}


void test_cxx11_tensor_cast_float16_cuda()
{
  CALL_SUBTEST(test_cuda_conversion());
  CALL_SUBTEST(test_fallback_conversion());
}
